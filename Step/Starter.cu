#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <cmath>
#include "ausmPlus.h"

using namespace std;

double gammma;
double mu;
double k;
double R;	

int main()
{
	gammma=1.4;
	mu=1.789*pow(10,-5);
	k=0.0257;
	R=286.9;
	
	double initial[4];
	double temp=300;
	double speed=1000;
	double speed_sound=sqrt(gammma*R*temp);
	double mach=speed/speed_sound;
	double pressure=101325/pow((1+(gammma-1)/2*mach*mach),gammma/(gammma-1));
	double density = pressure/(R*temp);
	double Re=(pressure/R/temp)*speed*13.85/mu;
	double Prandle=mu*1004/k;
	//Rho
	initial[0]=1;
	//Rho*U
	initial[1]=-1;
	//Rho *V
	initial[2]=0;
	//Rho*E, E is the internal energy including the kinetic energy(i.e. total intenal energy)
	initial[3]=1;
	//Time steps and delta_t
	double timesteps=750000;
	double deltat=0.01	;

	cout<<"pressure="<<pressure<<endl;
	cout<<" density="<<density<<endl;
	cout<<"static energy="<<R/(gammma-1)*temp*pow(101325/pressure,1/gammma)*initial[0]<<endl;
	cout<<"mach = "<<mach<<" speed = "<<speed<<endl;
	cout<<"Reynolds = "<<Re<<" Prandl = "<<Prandle<<endl;

	ausmplus(initial,timesteps,deltat,1,1,Re,Prandle,mach);

	return 0;
}