#include "hip/hip_runtime.h"
#include "ausmPlus.h"
#include <stdio.h>
	
__global__ void diffusiveFlux(cell *domain,double *R, double *gammma,double wall_temp, double prandle_inf, double M_inf, double Re)
{
	int x=blockIdx.x;
	int y=threadIdx.x;
	int note=-10;
	int faces=(int)domain[x].face[y]-1;
	int ourFlag=(int)domain[x].flag;
	double delu_delx=0.0,delv_delx=0.0,delu_dely=0.0,delv_dely=0.0;
	if(ourFlag==0 || ourFlag==4)
	{
		double x_cord[]={0,0},y_cord[]={0,0};
		
		if(faces<0 || faces>42860)
		{
			note=y;
		}

		int i1,i2;
		if(ourFlag==4 && y==note)
		{
			i1=note;
			i2=(note+1)%4;
			x_cord[1]=0.5*(domain[x].nodes[i1][0]+domain[x].nodes[i2][0]);
			y_cord[1]=0.5*(domain[x].nodes[i1][1]+domain[x].nodes[i2][1]);
		}

		for (int i = 0; i < 4; ++i)
		{
			if(ourFlag!=4 || (ourFlag==4 && y!=note))
			{
				//x_cordinate of the elements
				x_cord[0]+=0.25*(domain[x].nodes[i][0]);
				x_cord[1]+=0.25*(domain[faces].nodes[i][0]);
				//Y coordinate of the elements
				y_cord[0]+=0.25*(domain[x].nodes[i][1]);
				y_cord[1]+=0.25*(domain[faces].nodes[i][1]);
			}
			else
			{
				//x_cordinate of the elements
				x_cord[0]+=0.25*(domain[x].nodes[i][0]);
				//Y coordinate of the elements
				y_cord[0]+=0.25*(domain[x].nodes[i][1]);
			}
		}

		if(abs(x_cord[1]-x_cord[0])<=0.001)
		{
			delu_delx=0.0;
			delv_delx=0.0;
		}
		else
		{
			delu_delx=(domain[x].temp_var[y][1]/domain[x].temp_var[y][0]-domain[x].stateVar[1]/domain[x].stateVar[0])/(x_cord[1]-x_cord[0]);
			delv_delx=(domain[x].temp_var[y][2]/domain[x].temp_var[y][0]-domain[x].stateVar[2]/domain[x].stateVar[0])/(x_cord[1]-x_cord[0]);
		}
		if(abs(y_cord[1]-y_cord[0])<=0.001)
		{
			delu_dely=0.0;
			delv_dely=0.0;
		}
		else
		{
			delu_dely=(domain[x].temp_var[y][1]/domain[x].temp_var[y][0]-domain[x].stateVar[1]/domain[x].stateVar[0])/(y_cord[1]-y_cord[0]);
			delv_dely=(domain[x].temp_var[y][2]/domain[x].temp_var[y][0]-domain[x].stateVar[2]/domain[x].stateVar[0])/(y_cord[1]-y_cord[0]);
		}

		double tau_xx=2*(delu_delx-1/2*(delu_delx+delv_dely))*(*gammma*(*gammma-1)*M_inf*M_inf)/(Re*(1+*gammma*(*gammma-1)*M_inf*M_inf));
		double tau_yy=2*(delv_dely-1/2*(delu_delx+delv_dely))*(*gammma*(*gammma-1)*M_inf*M_inf)/(Re*(1+*gammma*(*gammma-1)*M_inf*M_inf));
		double tau_xy=(delu_dely+delv_delx)*(*gammma*(*gammma-1)*M_inf*M_inf)/(Re*(1+*gammma*(*gammma-1)*M_inf*M_inf));

		double temp[2];
		temp[0]=(gammma[0]-1)/R[0]*(domain[x].stateVar[3]-0.5*(pow(domain[x].stateVar[1],2)+pow(domain[x].stateVar[2],2))/domain[x].stateVar[0])/domain[x].stateVar[0];
		if(ourFlag!=4 || (ourFlag==4 && y!=note))
			temp[1]=(gammma[0]-1)/R[0]*(domain[x].temp_var[y][3]-0.5*(pow(domain[x].temp_var[y][1],2)\
				+pow(domain[x].temp_var[y][2],2))/domain[x].temp_var[y][0])/domain[x].temp_var[y][0];
		else
		{
			temp[1]=wall_temp;
		}

		double delT_delx,delT_dely;	
		if(abs(x_cord[1]-x_cord[0])<=0.001)
			delT_delx=0;
		else
			delT_delx=(temp[1]-temp[0])/(x_cord[1]-x_cord[0])*1/(prandle_inf*Re*(1+*gammma*(*gammma-1)*M_inf*M_inf));
		if(abs(y_cord[1]-y_cord[0])<=0.001)
			delT_dely=0;
		else
			delT_dely=(temp[1]-temp[0])/(y_cord[1]-y_cord[0])*1/(prandle_inf*Re*(1+*gammma*(*gammma-1)*M_inf*M_inf));

		double thetaX=(domain[x].stateVar[1]/domain[x].stateVar[0]*tau_xx+domain[x].stateVar[2]/domain[x].stateVar[0]*tau_xy+delT_delx);
		double thetaY=domain[x].stateVar[1]/domain[x].stateVar[0]*tau_xy+domain[x].stateVar[2]/domain[x].stateVar[0]*tau_yy+delT_dely;

		domain[x].diffflux[y][0]=0;
		domain[x].diffflux[y][1]=(tau_xx*domain[x].norms[y][0]+tau_xy*domain[x].norms[y][1])\
		*sqrt(pow(domain[x].nodes[y][0]-domain[x].nodes[(y+1)%4][0],2)+pow(domain[x].nodes[y][1]-domain[x].nodes[(y+1)%4][1],2));
		domain[x].diffflux[y][2]=(tau_xy*domain[x].norms[y][0]+tau_yy*domain[x].norms[y][1])\
		*sqrt(pow(domain[x].nodes[y][0]-domain[x].nodes[(y+1)%4][0],2)+pow(domain[x].nodes[y][1]-domain[x].nodes[(y+1)%4][1],2));
		domain[x].diffflux[y][3]=(thetaX*domain[x].norms[y][0]+thetaY*domain[x].norms[y][1])\
		*sqrt(pow(domain[x].nodes[y][0]-domain[x].nodes[(y+1)%4][0],2)+pow(domain[x].nodes[y][1]-domain[x].nodes[(y+1)%4][1],2));

		if(abs(0.25*(domain[x].nodes[0][0]+domain[x].nodes[1][0]+domain[x].nodes[2][0]+domain[x].nodes[3][0])-0.3748442233)<0.000001 && abs(0.25*(domain[x].nodes[0][1]+domain[x].nodes[1][1]+domain[x].nodes[2][1]+domain[x].nodes[3][1])-21.6161422729)<0.00001)
		{
			printf("diffusion flux %5.14lf %5.14lf %5.14lf %5.14lf\n",domain[x].diffflux[y][0],domain[x].diffflux[y][1],domain[x].diffflux[y][2],domain[x].diffflux[y][3]);
		}
	
	}
}	